#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "Array.hpp"
/* This code will multiply a matrix by a vector and
   check the result.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

/************************/
/* TEST KERNEL FUNCTION */
/************************/
__global__ void MyKernel(int *a, int *b, int *c, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

/********/
/* MAIN */
/********/
int main() {
    const int N = 1000000;

    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the maximum
                     // occupancy for a full device launch
    int gridSize;    // The actual grid size needed, based on input size

    int *h_vec1 = (int *)malloc(N * sizeof(int));
    int *h_vec2 = (int *)malloc(N * sizeof(int));
    int *h_vec3 = (int *)malloc(N * sizeof(int));
    int *h_vec4 = (int *)malloc(N * sizeof(int));

    int *d_vec1;
    hipMalloc((void **)&d_vec1, N * sizeof(int));
    int *d_vec2;
    hipMalloc((void **)&d_vec2, N * sizeof(int));
    int *d_vec3;
    hipMalloc((void **)&d_vec3, N * sizeof(int));

    for (int i = 0; i < N; i++) {
        h_vec1[i] = 10;
        h_vec2[i] = 20;
        h_vec4[i] = h_vec1[i] + h_vec2[i];
    }

    hipMemcpy(d_vec1, h_vec1, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, h_vec2, N * sizeof(int), hipMemcpyHostToDevice);

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, MyKernel, 0,
                                       N);

    // Round up according to array size
    gridSize = (N + blockSize - 1) / blockSize;

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Occupancy calculator elapsed time:  %3.3f ms \n", time);

    hipEventRecord(start, 0);

    MyKernel<<<gridSize, blockSize>>>(d_vec1, d_vec2, d_vec3, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Kernel elapsed time:  %3.3f ms \n", time);

    printf("Blocksize %i\n", blockSize);

    hipMemcpy(h_vec3, d_vec3, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        if (h_vec3[i] != h_vec4[i]) {
            printf("Error at i = %i! Host = %i; Device = %i\n", i, h_vec4[i],
                   h_vec3[i]);
            return;
        };
    }

    printf("Test passed\n");
}
