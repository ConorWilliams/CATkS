#include "hip/hip_runtime.h"
#include <iostream>

#include "Array.hpp"

#include "EAM.hpp"

struct A {};
struct B : public A {};

template <typename T, size_t N> int foo(Array_d<T, N>) { return 3; }
int foo(A) { return 1; }

template <typename T, size_t N> __global__ void MyKernel(Array_d<T, N> arr) {
    arr(threadIdx.x, threadIdx.y) *= max(2, (int)arr(threadIdx.x, threadIdx.y));
}

int main() {

    // ArrayGPU<float, 2> arrG;

    constexpr std::size_t n = 46;

    Array_h<float, 2> arr(n, n / 2);

    for (std::size_t j = 0; j < n / 2; ++j) {
        for (std::size_t i = 0; i < n; ++i) {
            arr(i, j) = i + n * j;
        }
    }

    for (std::size_t i = 0; i < n; ++i) {
        for (std::size_t j = 0; j < n / 2; ++j) {
            std::cout << arr(i, j) << ' ';
        }
        std::cout << '\n';
    }

    arr.hostToDevice();

    MyKernel<<<1, {n, n / 2}>>>(arr);

    arr.deviceToHost();

    hipDeviceSynchronize();

    // arrG(1, 1) = arrC(2, 2);

    std::cout << "passing " << foo(B{}) << '\n';
    std::cout << "passing " << foo(arr) << '\n';

    for (std::size_t i = 0; i < n; ++i) {
        for (std::size_t j = 0; j < n / 2; ++j) {
            std::cout << arr(i, j) << ' ';
        }
        std::cout << '\n';
    }

    std::cout << "ALL GOOD\n";

    return 0;
}
